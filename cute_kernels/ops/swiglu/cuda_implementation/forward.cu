#include "hip/hip_runtime.h"
// **************************************************
// Copyright (c) 2025, Mayank Mishra
// **************************************************

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using uint32 = ck::uint32;
using uint64 = ck::uint64;

template <typename scalar_t>
inline __device__ scalar_t _swiglu_forward(const scalar_t &gate, const scalar_t &up) {
    using dtype = ck::DType<scalar_t>;

    fp32 _up = dtype::upcast(up);
    fp32 _gate = dtype::upcast(gate);
    fp32 _sigmoid = ck::sigmoid<fp32, fp32>(_gate);

    _sigmoid *= _gate * _up;

    return dtype::downcast(_sigmoid);
}

template <typename scalar_t>
__global__ void swiglu_forward_cuda_kernel(const scalar_t *gate,
                                           const scalar_t *up,
                                           scalar_t *output,
                                           const uint64 N) {
    constexpr uint32 N_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();

    const uint32 thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32 N_vec = N / N_per_thread;

    if (thread_id < N_vec) {
        const scalar_t *gate_vec = ck_mem::load_128_bits<scalar_t>(gate, thread_id);
        const scalar_t *up_vec = ck_mem::load_128_bits<scalar_t>(up, thread_id);
        scalar_t output_buffer[N_per_thread];

        for (uint32 i = 0; i < N_per_thread; i++) {
            output_buffer[i] = _swiglu_forward<scalar_t>(gate_vec[i], up_vec[i]);
        }

        ck_mem::store_128_bits<scalar_t>(output_buffer, output, thread_id);
    }

    const uint32 index = N_vec * N_per_thread + thread_id;
    if (index < N) {
        output[index] = _swiglu_forward<scalar_t>(gate[index], up[index]);
    }
}

void swiglu_forward_cuda(const torch::Tensor &gate,
                         const torch::Tensor &up,
                         torch::Tensor &output,
                         const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(gate);
    CHECK_CUDA_TENSOR(up);
    CHECK_CUDA_TENSOR(output);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    const uint64 total_elements = gate.numel();

    DISPATCH_FLOAT_KERNEL(gate.scalar_type(), "swiglu_forward_cuda_kernel", scalar_t, ([&] {
                              const uint32 N_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();
                              const uint32 N_per_block = BLOCK_SIZE * N_per_thread;

                              std::vector<ck::ChunkedArray<scalar_t>> gate_chunks =
                                  ck::chunk_array<scalar_t>(gate.data_ptr<scalar_t>(), total_elements);
                              std::vector<ck::ChunkedArray<scalar_t>> up_chunks =
                                  ck::chunk_array<scalar_t>(up.data_ptr<scalar_t>(), total_elements);
                              std::vector<ck::ChunkedArray<scalar_t>> output_chunks =
                                  ck::chunk_array<scalar_t>(output.data_ptr<scalar_t>(), total_elements);

                              for (uint32 i = 0; i < gate_chunks.size(); i++) {
                                  ck::ChunkedArray<scalar_t> gate_chunk = gate_chunks[i];
                                  ck::ChunkedArray<scalar_t> up_chunk = up_chunks[i];
                                  ck::ChunkedArray<scalar_t> output_chunk = output_chunks[i];

                                  const uint64 N = gate_chunk.num_elements;
                                  const uint32 NUM_BLOCKS = ck::ceil_divide<uint64>(N, N_per_block);

                                  swiglu_forward_cuda_kernel<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE>>>(
                                      gate_chunk.array, up_chunk.array, output_chunk.array, N);
                              }
                          }));
}
