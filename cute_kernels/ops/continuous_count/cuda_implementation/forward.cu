#include "hip/hip_runtime.h"
// **************************************************
// Copyright (c) 2025, Mayank Mishra
// **************************************************

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/cute_kernels.h"

#define MAX_ALLOWED_E 16384

namespace cg = cooperative_groups;
namespace ck = cute_kernels;
namespace ck_mem = ck::memory;

using int32 = ck::int32;
using int64 = ck::int64;
using uint32 = ck::uint32;
using uint64 = ck::uint64;

inline __device__ void _looped_atomic_add(uint32 *source, uint32 *destination, const uint32 &E) {
    uint32 index = threadIdx.x;
    while (index < E) {
        atomicAdd(&destination[index], source[index]);
        index += blockDim.x;
    }
}

inline __device__ void _initialize_global_output(uint32 *output,
                                                 const uint32 &E,
                                                 const uint32 &global_thread_id,
                                                 const uint32 &total_threads) {
    const uint32 E4 = E >> 2;

    uint32 init_value[] = {0, 0, 0, 0};

    for (uint32 i = global_thread_id; i < E4; i += total_threads) {
        ck_mem::store_128_bits<uint32>(init_value, output, i);
    }

    const uint32 index = (E4 << 2) + global_thread_id;
    if (index < E) {
        output[index] = 0;
    }
}

template <typename scalar_t>
inline __device__ void _update_local_count(const scalar_t *x,
                                           uint32 *shared_memory,
                                           const uint64 &N,
                                           const uint32 &global_thread_id,
                                           const uint32 &total_threads) {
    constexpr uint32 N_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();
    const uint32 N_vec = N / N_per_thread;

    for (uint32 i = global_thread_id; i < N_vec; i += total_threads) {
        const scalar_t *x_vec = ck_mem::load_128_bits<scalar_t>(x, i);

        for (uint32 j = 0; j < N_per_thread; j++) {
            atomicAdd(&shared_memory[x_vec[j]], 1);
        }
    }

    const uint32 index = (N_vec * N_per_thread) + global_thread_id;
    if (index < N) {
        atomicAdd(&shared_memory[x[index]], 1);
    }
}

template <typename scalar_t>
inline __device__ uint32 *_get_shared_memory(const uint32 &E) {
    extern __shared__ uint32 shared_memory[];

    uint32 index = threadIdx.x;
    while (index < E) {
        shared_memory[index] = 0;
        index += blockDim.x;
    }

    return shared_memory;
}

template <typename scalar_t, bool do_sort>
__global__ void continuous_count_cuda_kernel(const scalar_t *x,
                                             uint32 *output,
                                             scalar_t *sorted_output,
                                             uint64 *sorted_indices,
                                             const uint64 N,
                                             const uint32 E) {
    uint32 *shared_memory = _get_shared_memory<scalar_t>(E);

    const uint32 global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32 grid_size = gridDim.x * blockDim.x;

    _initialize_global_output(output, E, global_thread_id, grid_size);
    cg::this_grid().sync();

    _update_local_count<scalar_t>(x, shared_memory, N, global_thread_id, grid_size);

    cg::cluster_group cluster = cg::this_cluster();
    const bool is_first_cluster_block = cluster.block_rank() == 0;

    __syncthreads();

    if (!is_first_cluster_block) {
        _looped_atomic_add(shared_memory, cluster.map_shared_rank(shared_memory, 0), E);
    }

    cluster.sync();

    // write the output to the global memory
    if (is_first_cluster_block) {
        _looped_atomic_add(shared_memory, output, E);
    }
}

void continuous_count_cuda(const torch::Tensor &x,
                           torch::Tensor &output,
                           std::optional<torch::Tensor> &_sorted_output,
                           std::optional<torch::Tensor> &_sorted_indices,
                           const uint32 &E,
                           const uint32 &THREAD_BLOCK_CLUSTER_SIZE,
                           const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(x);
    CHECK_CUDA_TENSOR(output);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    TORCH_CHECK(E <= MAX_ALLOWED_E);

    const uint64 N = x.numel();
    CHECK_WITHIN_UINT32(N);

    const bool do_sort = _sorted_output.has_value();

    TORCH_CHECK(_sorted_indices.has_value() == do_sort);

    const uint32 num_SMs = ck::get_num_SMs();
    const uint32 max_num_blocks = ck::get_max_thread_blocks(num_SMs, THREAD_BLOCK_CLUSTER_SIZE);

    DISPATCH_INT_KERNEL(x.scalar_type(), "continuous_count_cuda_kernel", scalar_t, ([&] {
                            if (do_sort) {
                                hipFuncSetAttribute(reinterpret_cast<const void*>(continuous_count_cuda_kernel<scalar_t), true>,
                                                     hipFuncAttributeMaxDynamicSharedMemorySize,
                                                     MAX_ALLOWED_E * sizeof(uint32));
                            } else {
                                hipFuncSetAttribute(reinterpret_cast<const void*>(continuous_count_cuda_kernel<scalar_t), false>,
                                                     hipFuncAttributeMaxDynamicSharedMemorySize,
                                                     MAX_ALLOWED_E * sizeof(uint32));
                            }

                            auto [NUM_BLOCKS, cluster_size] =
                                ck::get_num_blocks(N, BLOCK_SIZE, max_num_blocks, THREAD_BLOCK_CLUSTER_SIZE);

                            // dynamically sized clusters need this stupid way of launching the kernel
                            hipLaunchConfig_t launch_config = {0};
                            launch_config.blockDim = BLOCK_SIZE;
                            launch_config.gridDim = NUM_BLOCKS;
                            launch_config.dynamicSmemBytes = E * sizeof(uint32);

                            hipLaunchAttribute attributes[2];

                            attributes[0].id = cudaLaunchAttributeClusterDimension;
                            attributes[0].val.clusterDim.x = cluster_size;
                            attributes[0].val.clusterDim.y = 1;
                            attributes[0].val.clusterDim.z = 1;

                            attributes[1].id = hipLaunchAttributeCooperative;
                            attributes[1].val.cooperative = 1;

                            launch_config.attrs = attributes;
                            launch_config.numAttrs = 2;

                            cudaLaunchKernelEx(&launch_config,
                                               do_sort ? continuous_count_cuda_kernel<scalar_t, true>
                                                       : continuous_count_cuda_kernel<scalar_t, false>,
                                               x.data_ptr<scalar_t>(),
                                               output.data_ptr<uint32>(),
                                               do_sort ? _sorted_output.value().data_ptr<scalar_t>() : nullptr,
                                               do_sort ? _sorted_indices.value().data_ptr<uint64>() : nullptr,
                                               N,
                                               E);
                        }));
}
